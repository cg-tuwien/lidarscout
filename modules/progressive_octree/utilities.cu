#include "hip/hip_runtime.h"
// Some code in this file, particularly frustum, ray and intersection tests,
// is adapted from three.js. Three.js is licensed under the MIT license
// This file this follows the three.js licensing
// License: MIT https://github.com/mrdoob/three.js/blob/dev/LICENSE

#define CUB_DISABLE_BF16_SUPPORT

#define GLM_FORCE_CUDA
#define GLM_ENABLE_EXPERIMENTAL
#define CUDA_VERSION 12000

namespace std{
	using size_t = ::size_t;
};

#include "./libs/glm/glm/glm.hpp"
#include <hip/hip_cooperative_groups.h>
#include "HostDeviceInterface.h"

using namespace std;
namespace cg = cooperative_groups;


extern "C" __global__ 
void kernel_denormalize(
	float* heightmap,
	int width, int height,
	float patchRadiusDenormalization,
	float tileCenterZ
){

	int texelID = cg::this_grid().thread_rank();

	if(texelID >= width * height) return;

	float heightmapNumericalStabilityFactor = 10.0f;

	float normalized = heightmap[texelID];

	heightmap[texelID] = normalized / heightmapNumericalStabilityFactor * patchRadiusDenormalization + tileCenterZ;
					
}

extern "C" __global__ 
void kernel_colors_vec4_to_uint32(
	vec3* source, int* mask,
	uint32_t* target,
	int width, int height
){

	int texelID = cg::this_grid().thread_rank();

	if(texelID >= width * height) return;

	int x = texelID % 64;
	int y = texelID / 64;

	uint32_t color = 0;
	uint8_t* rgba = (uint8_t*)&color;
	rgba[0] = 255.0f * source[texelID].r;
	rgba[1] = 255.0f * source[texelID].g;
	rgba[2] = 255.0f * source[texelID].b;
	rgba[3] = 255;

	int sourceMaskTexelIndex = (y + 16) * 96 + (x + 16);
	int maskValue = mask[sourceMaskTexelIndex];

	if(maskValue < 0){
		color = 0;
	}

	for(int ox = -1; ox <= 1; ox++)
	for(int oy = -1; oy <= 1; oy++)
	{
		int px = x + ox;
		int py = y + oy;

		if(px < 0 || px >= 64) continue;
		if(py < 0 || py >= 64) continue;

		int sourceMaskTexelIndex = (py + 16) * 96 + (px + 16);
		int maskValue = mask[sourceMaskTexelIndex];

		if(maskValue < 0){
			color = 0;
		}
	}

	// TODO: remove texels without points in vicinity

	// uint32_t color = 0xff00ff00;
	target[texelID] = color;
}


extern "C" __global__ 
void kernel_colors_vec4_to_uint64(
	vec3* source, int* mask,
	uint64_t* target,
	int width, int height
){

	int texelID = cg::this_grid().thread_rank();

	if(texelID >= width * height) return;

	int x = texelID % 64;
	int y = texelID / 64;

	uint64_t color = 0;
	uint16_t* rgba = (uint16_t*)&color;
	rgba[0] = 255.0f * source[texelID].r;
	rgba[1] = 255.0f * source[texelID].g;
	rgba[2] = 255.0f * source[texelID].b;
	rgba[3] = 1;

	int sourceMaskTexelIndex = (y + 16) * 96 + (x + 16);
	int maskValue = mask[sourceMaskTexelIndex];

	if(maskValue < 0){
		color = 0;
	}

	// for(int ox = -3; ox <= 3; ox++)
	// for(int oy = -3; oy <= 3; oy++)
	// {
	// 	int px = x + ox;
	// 	int py = y + oy;

	// 	if(px < 0 || px >= 64) continue;
	// 	if(py < 0 || py >= 64) continue;

	// 	int sourceMaskTexelIndex = (py + 16) * 96 + (px + 16);
	// 	int maskValue = mask[sourceMaskTexelIndex];

	// 	if(maskValue < 0){
	// 		color = 0;
	// 	}
	// }
	
	// uint32_t color = 0xff00ff00;
	target[texelID] = color;
}