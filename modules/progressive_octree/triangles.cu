#include "hip/hip_runtime.h"
#define CUB_DISABLE_BF16_SUPPORT

#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "./HostDeviceInterface_triangles.h"
#include "./structures.cuh"
#include "./utils.cuh"

constexpr int MAX_VERYLARGE_TRIANGLES = 10 * 1024;
__device__ uint32_t numProcessedTriangles;
__device__ uint32_t veryLargeTriangleIndices[MAX_VERYLARGE_TRIANGLES];
__device__ uint32_t veryLargeTriangleCounter;

inline float4 toScreenCoord(float3 p, mat4& transform, int width, int height) {
	float4 pos = transform * float4{p.x, p.y, p.z, 1.0f};

	pos.x = pos.x / pos.w;
	pos.y = pos.y / pos.w;

	return float4{(pos.x * 0.5f + 0.5f) * width, (pos.y * 0.5f + 0.5f) * height, pos.z, pos.w};
}

inline uint32_t computeColor(
		int triangleIndex,
		TriangleData triangles,
		TriangleMaterial material,
		CTexture texture,
		float s,
		float t,
		float v) {

	uint32_t color;
	uint8_t* rgb = (uint8_t*)&color;

	color = triangleIndex * 123456;
	color = 0x0000ff00;

	material.mode = MATERIAL_MODE_UVS;

	if (material.mode == MATERIAL_MODE_COLOR) {
		rgb[0] = 255.0f * material.color.x;
		rgb[1] = 255.0f * material.color.y;
		rgb[2] = 255.0f * material.color.z;
		rgb[3] = 255.0f * material.color.w;
	} else if (material.mode == MATERIAL_MODE_VERTEXCOLOR && triangles.colors != nullptr) {
		uint8_t rgba_0[4];
		uint8_t rgba_1[4];
		uint8_t rgba_2[4];
		memcpy(rgba_0, &triangles.colors[3 * triangleIndex + 0], 4);
		memcpy(rgba_1, &triangles.colors[3 * triangleIndex + 1], 4);
		memcpy(rgba_2, &triangles.colors[3 * triangleIndex + 2], 4);

		float3 c0 = {rgba_0[0], rgba_0[1], rgba_0[2]};
		float3 c1 = {rgba_1[0], rgba_1[1], rgba_1[2]};
		float3 c2 = {rgba_2[0], rgba_2[1], rgba_2[2]};

		float3 c = v * c0 + s * c1 + t * c2;
		color = (int(c.x) << 0) | (int(c.y) << 8) | (int(c.z) << 16);

	} else if (material.mode == MATERIAL_MODE_UVS && triangles.uv != nullptr) {
		uint8_t rgba_0[4];
		uint8_t rgba_1[4];
		uint8_t rgba_2[4];

		float2 uv0 = {
				triangles.uv[3 * triangleIndex + 0].x,
				triangles.uv[3 * triangleIndex + 0].y,
		};
		float2 uv1 = {
				triangles.uv[3 * triangleIndex + 1].x,
				triangles.uv[3 * triangleIndex + 1].y,
		};
		float2 uv2 = {
				triangles.uv[3 * triangleIndex + 2].x,
				triangles.uv[3 * triangleIndex + 2].y,
		};

		float2 uv = v * uv0 + s * uv1 + t * uv2;

		uv = uv / material.uv_scale + material.uv_offset;

		uv.x = clamp(uv.x, 0.0f, 1.0f);
		uv.y = clamp(uv.y, 0.0f, 1.0f);

		if (texture.data) {
			auto sampleTexture = [&](float2 uv, CTexture texture) {
				int tx = int(uv.x * texture.width) % texture.width;
				int ty = int(uv.y * texture.height) % texture.height;

				int texelID = tx + texture.width * ty;

				if (texelID < 0) {
					printf("uv:	%.2f, %.2f\n", uv.x, uv.y);
					printf("texture %d, %d\n", texture.width, texture.height);
					printf("test %d\n", texelID);
				}

				if (texelID < 0)
					return 0xff0000ff;

				uint32_t r = texture.data[4 * texelID + 0];
				uint32_t g = texture.data[4 * texelID + 1];
				uint32_t b = texture.data[4 * texelID + 2];
				uint32_t a = texture.data[4 * texelID + 3];

				uint32_t color = (r << 0) | (g << 8) | (b << 16) | (a << 24);

				return color;
			};

			color = sampleTexture(uv, texture);
		} else {
			rgb[0] = 255.0f * uv.x;
			rgb[1] = 255.0f * uv.y;
		}
	} else {
		color = 0xff0000ff;
	}

	return color;
}

inline void rasterizeVeryLargeTriangles(
		TriangleData triangles,
		TriangleMaterial material,
		CTexture texture,
		RenderTarget target,
		mat4& transform) {

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	for (int i = 0; i < min(veryLargeTriangleCounter, MAX_VERYLARGE_TRIANGLES); i++) {
		int triangleIndex = veryLargeTriangleIndices[i];

		float3 v0 = triangles.position[3 * triangleIndex + 0];
		float3 v1 = triangles.position[3 * triangleIndex + 1];
		float3 v2 = triangles.position[3 * triangleIndex + 2];

		float4 p0 = toScreenCoord(v0, transform, target.width, target.height);
		float4 p1 = toScreenCoord(v1, transform, target.width, target.height);
		float4 p2 = toScreenCoord(v2, transform, target.width, target.height);

		if (p0.w < 0.0 || p1.w < 0.0 || p2.w < 0.0) {
			continue;
		}

		float2 v01 = {p1.x - p0.x, p1.y - p0.y};
		float2 v02 = {p2.x - p0.x, p2.y - p0.y};

		auto cross = [](float2 a, float2 b) {
			return a.x * b.y - a.y * b.x;
		};

		{	// backface culling
			float w = cross(v01, v02);
			if (w < 0.0)
				continue;
		}

		// compute screen-space bounding rectangle
		float min_x = min(min(p0.x, p1.x), p2.x);
		float min_y = min(min(p0.y, p1.y), p2.y);
		float max_x = max(max(p0.x, p1.x), p2.x);
		float max_y = max(max(p0.y, p1.y), p2.y);

		// clamp to screen
		min_x = clamp(min_x, 0.0f, (float)target.width);
		min_y = clamp(min_y, 0.0f, (float)target.height);
		max_x = clamp(max_x, 0.0f, (float)target.width);
		max_y = clamp(max_y, 0.0f, (float)target.height);

		int size_x = ceil(max_x) - floor(min_x);
		int size_y = ceil(max_y) - floor(min_y);
		int numFragments = size_x * size_y;

		int fragsPerBlock = numFragments / (grid.num_blocks() - 1) + 1;

		int startFrag = (grid.block_rank() + 0) * fragsPerBlock;
		int endFrag = (grid.block_rank() + 1) * fragsPerBlock;

		endFrag = min(endFrag, numFragments);

		int numProcessedSamples = 0;
		for (int fragOffset = startFrag; fragOffset < endFrag; fragOffset += block.num_threads()) {
			// safety mechanism: don't draw more than <x> pixels per thread
			if (numProcessedSamples > 10'000) {
				break;
			}

			int fragID = fragOffset + block.thread_rank();
			int fragX = fragID % size_x;
			int fragY = fragID / size_x;

			float2 pFrag = {floor(min_x) + float(fragX), floor(min_y) + float(fragY)};
			float2 sample = {pFrag.x - p0.x, pFrag.y - p0.y};

			// v: vertex[0], s: vertex[1], t: vertex[2]
			float s = cross(sample, v02) / cross(v01, v02);
			float t = cross(v01, sample) / cross(v01, v02);
			float v = 1.0f - (s + t);

			int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
			int pixelID = pixelCoords.x + pixelCoords.y * target.width;
			pixelID = clamp(pixelID, 0, int(target.width * target.height) - 1);

			if (s >= 0.0f) {
				if (t >= 0.0f) {
					if (v >= 0.0f) {
						uint32_t color = computeColor(triangleIndex, triangles, material, texture, s, t, v);

						float depth = v * p0.w + s * p1.w + t * p2.w;
						uint64_t udepth = *((uint32_t*)&depth);
						uint64_t pixel = (udepth << 32ull) | color;

						atomicMin(&target.framebuffer[pixelID], pixel);
					}
				}
			}

			numProcessedSamples++;
		}
	}
}

inline void rasterizeLargeTriangles(
		TriangleData triangles,
		TriangleMaterial material,
		CTexture texture,
		RenderTarget target,
		int* triangleIndices,
		uint32_t numTriangles,
		mat4& transform) {

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	for (int i = 0; i < numTriangles; i++) {

		int triangleIndex = triangleIndices[i];

		float3 v0 = triangles.position[3 * triangleIndex + 0];
		float3 v1 = triangles.position[3 * triangleIndex + 1];
		float3 v2 = triangles.position[3 * triangleIndex + 2];

		float4 p0 = toScreenCoord(v0, transform, target.width, target.height);
		float4 p1 = toScreenCoord(v1, transform, target.width, target.height);
		float4 p2 = toScreenCoord(v2, transform, target.width, target.height);

		// cull a triangle if one of its vertices is closer than depth 0
		if (p0.w < 0.0 || p1.w < 0.0 || p2.w < 0.0) {
			continue;
		}

		float2 v01 = {p1.x - p0.x, p1.y - p0.y};
		float2 v02 = {p2.x - p0.x, p2.y - p0.y};

		auto cross = [](float2 a, float2 b) {
			return a.x * b.y - a.y * b.x;
		};

		{	// backface culling
			float w = cross(v01, v02);
			if (w < 0.0)
				continue;
		}

		// compute screen-space bounding rectangle
		float min_x = min(min(p0.x, p1.x), p2.x);
		float min_y = min(min(p0.y, p1.y), p2.y);
		float max_x = max(max(p0.x, p1.x), p2.x);
		float max_y = max(max(p0.y, p1.y), p2.y);

		// clamp to screen
		min_x = clamp(min_x, 0.0f, (float)target.width);
		min_y = clamp(min_y, 0.0f, (float)target.height);
		max_x = clamp(max_x, 0.0f, (float)target.width);
		max_y = clamp(max_y, 0.0f, (float)target.height);

		int size_x = ceil(max_x) - floor(min_x);
		int size_y = ceil(max_y) - floor(min_y);
		int numFragments = size_x * size_y;

		// iterate through fragments in bounding rectangle and draw if within triangle
		int numProcessedSamples = 0;
		for (int fragOffset = 0; fragOffset < numFragments; fragOffset += block.num_threads()) {

			// safety mechanism: don't draw more than <x> pixels per thread
			if (numProcessedSamples > 10'000)
				break;

			int fragID = fragOffset + block.thread_rank();
			int fragX = fragID % size_x;
			int fragY = fragID / size_x;

			float2 pFrag = {floor(min_x) + float(fragX), floor(min_y) + float(fragY)};
			float2 sample = {pFrag.x - p0.x, pFrag.y - p0.y};

			// v: vertex[0], s: vertex[1], t: vertex[2]
			float s = cross(sample, v02) / cross(v01, v02);
			float t = cross(v01, sample) / cross(v01, v02);
			float v = 1.0 - (s + t);

			int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
			int pixelID = pixelCoords.x + pixelCoords.y * target.width;
			pixelID = clamp(pixelID, 0, int(target.width * target.height) - 1);

			if (s >= 0.0f) {
				if (t >= 0.0f) {
					if (v >= 0.0f) {
						uint32_t color = computeColor(triangleIndex, triangles, material, texture, s, t, v);

						float depth = v * p0.w + s * p1.w + t * p2.w;
						uint64_t udepth = *((uint32_t*)&depth);
						uint64_t pixel = (udepth << 32ull) | color;

						atomicMin(&target.framebuffer[pixelID], pixel);
					}
				}
			}

			numProcessedSamples++;
		}
	}
}

void rasterizeTriangles(TriangleData triangles, TriangleMaterial material, CTexture texture, RenderTarget target) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	if (grid.thread_rank() == 0) {
		numProcessedTriangles = 0;
		veryLargeTriangleCounter = 0;
	}

	grid.sync();

	mat4 transform = target.proj * target.view * triangles.transform;

	constexpr int TRIANGLES_PER_SWEEP = 32;
	__shared__ float3 sh_positions[3 * TRIANGLES_PER_SWEEP];
	__shared__ float2 sh_uvs[3 * TRIANGLES_PER_SWEEP];
	__shared__ int sh_blockTriangleOffset;

	__shared__ struct {
		int numTriangles;
		int indices[TRIANGLES_PER_SWEEP];
	} largeTriangleSchedule;

	while (true) {
		block.sync();
		if (block.thread_rank() == 0) {
			sh_blockTriangleOffset = atomicAdd(&numProcessedTriangles, TRIANGLES_PER_SWEEP);
			largeTriangleSchedule.numTriangles = 0;
		}
		block.sync();

		int numTrianglesInBlock = min(int(triangles.count) - sh_blockTriangleOffset, TRIANGLES_PER_SWEEP);

		if (numTrianglesInBlock <= 0)
			break;

		// load triangles into shared memory
		for (int i = block.thread_rank(); i < numTrianglesInBlock; i += block.size()) {
			int triangleIndex = sh_blockTriangleOffset + i;
			sh_positions[3 * i + 0] = triangles.position[3 * triangleIndex + 0];
			sh_positions[3 * i + 1] = triangles.position[3 * triangleIndex + 1];
			sh_positions[3 * i + 2] = triangles.position[3 * triangleIndex + 2];

			sh_uvs[3 * i + 0] = triangles.uv[3 * triangleIndex + 0];
			sh_uvs[3 * i + 1] = triangles.uv[3 * triangleIndex + 1];
			sh_uvs[3 * i + 2] = triangles.uv[3 * triangleIndex + 2];
		}

		block.sync();

		// draw triangles
		for (int i = block.thread_rank(); i < numTrianglesInBlock; i += block.size()) {
			int triangleIndex = sh_blockTriangleOffset + i;

			float3 v_0 = sh_positions[3 * i + 0];
			float3 v_1 = sh_positions[3 * i + 1];
			float3 v_2 = sh_positions[3 * i + 2];

			float4 p_0 = toScreenCoord(v_0, transform, target.width, target.height);
			float4 p_1 = toScreenCoord(v_1, transform, target.width, target.height);
			float4 p_2 = toScreenCoord(v_2, transform, target.width, target.height);

			if (p_0.w < 0.0f || p_1.w < 0.0f || p_2.w < 0.0f)
				continue;

			float2 v_01 = {p_1.x - p_0.x, p_1.y - p_0.y};
			float2 v_02 = {p_2.x - p_0.x, p_2.y - p_0.y};

			auto cross = [](float2 a, float2 b) {
				return a.x * b.y - a.y * b.x;
			};

			{	// backface culling
				float w = cross(v_01, v_02);
				if (w < 0.0)
					continue;
			}

			// compute screen-space bounding rectangle
			float min_x = min(min(p_0.x, p_1.x), p_2.x);
			float min_y = min(min(p_0.y, p_1.y), p_2.y);
			float max_x = max(max(p_0.x, p_1.x), p_2.x);
			float max_y = max(max(p_0.y, p_1.y), p_2.y);

			// clamp to screen
			min_x = clamp(min_x, 0.0f, (float)target.width);
			min_y = clamp(min_y, 0.0f, (float)target.height);
			max_x = clamp(max_x, 0.0f, (float)target.width);
			max_y = clamp(max_y, 0.0f, (float)target.height);

			int size_x = ceil(max_x) - floor(min_x);
			int size_y = ceil(max_y) - floor(min_y);
			int numFragments = size_x * size_y;

			if (numFragments > 40'000) {
				uint32_t index = atomicAdd(&veryLargeTriangleCounter, 1);
				veryLargeTriangleIndices[index] = triangleIndex;
				continue;
			} else if (numFragments > 1024) {
				// TODO: schedule block-wise rasterization
				uint32_t index = atomicAdd(&largeTriangleSchedule.numTriangles, 1);
				largeTriangleSchedule.indices[index] = triangleIndex;
				continue;
			}

			int numProcessedSamples = 0;
			for (int fragOffset = 0; fragOffset < numFragments; fragOffset += 1) {

				// safety mechanism: don't draw more than <x> pixels per thread
				if (numProcessedSamples > 2000)
					break;

				int fragID = fragOffset;	// + block.thread_rank();
				int fragX = fragID % size_x;
				int fragY = fragID / size_x;

				float2 pFrag = {floor(min_x) + float(fragX), floor(min_y) + float(fragY)};
				float2 sample = {pFrag.x - p_0.x, pFrag.y - p_0.y};

				// v: vertex[0], s: vertex[1], t: vertex[2]
				float s = cross(sample, v_02) / cross(v_01, v_02);
				float t = cross(v_01, sample) / cross(v_01, v_02);
				float v = 1.0f - (s + t);

				int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
				int pixelID = pixelCoords.x + pixelCoords.y * target.width;
				pixelID = clamp(pixelID, 0, int(target.width * target.height) - 1);

				if (s >= 0.0f) {
					if (t >= 0.0f) {
						if (v >= 0.0f) {
							uint32_t color = computeColor(triangleIndex, triangles, material, texture, s, t, v);

							float depth = v * p_0.w + s * p_1.w + t * p_2.w;
							uint64_t udepth = *((uint32_t*)&depth);
							uint64_t pixel = (udepth << 32ull) | color;

							atomicMin(&target.framebuffer[pixelID], pixel);
						}
					}
				}

				++numProcessedSamples;
			}
		}

		block.sync();

		// do blockwise rasterization for triangles that were too large for thread-wise rasterization
		rasterizeLargeTriangles(
				triangles,
				material,
				texture,
				target,
				largeTriangleSchedule.indices,
				largeTriangleSchedule.numTriangles,
				transform);
	}

	grid.sync();
}

extern "C" __global__ void
kernel_drawTriangles(TriangleData triangles, TriangleMaterial material, CTexture texture, RenderTarget target) {
	auto block = cg::this_thread_block();
	material.mode = MATERIAL_MODE_UVS;
	rasterizeTriangles(triangles, material, texture, target);
}

uint32_t colorFromIndex(uint32_t index) {
	return ((index % 127) + 1) * 12345678;
}

template <typename VertexShader>
inline void rasterizeLargeTrianglesInstanced(
		TriangleData triangles,
		TriangleMaterial material,
		CTexture texture,
		RenderTarget target,
		int* triangleIndices,
		uint32_t numTriangles,
		mat4& transform,
		VertexShader&& vertexShader) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	for (int i = 0; i < numTriangles; ++i) {
		int instanceIndex = triangleIndices[i] / triangles.count;
		int triangleIndex = triangleIndices[i] % triangles.count;

		float3 v0 = vertexShader(3 * triangleIndex + 0, instanceIndex);
		float3 v1 = vertexShader(3 * triangleIndex + 1, instanceIndex);
		float3 v2 = vertexShader(3 * triangleIndex + 2, instanceIndex);

		float4 p0 = toScreenCoord(v0, transform, target.width, target.height);
		float4 p1 = toScreenCoord(v1, transform, target.width, target.height);
		float4 p2 = toScreenCoord(v2, transform, target.width, target.height);

		// cull a triangle if one of its vertices is closer than depth 0
		if (p0.w < 0.0 || p1.w < 0.0 || p2.w < 0.0) {
			continue;
		}

		float2 v01 = {p1.x - p0.x, p1.y - p0.y};
		float2 v02 = {p2.x - p0.x, p2.y - p0.y};

		auto cross = [](float2 a, float2 b) {
			return a.x * b.y - a.y * b.x;
		};

		{	// backface culling
			float w = cross(v01, v02);
			if (w < 0.0)
				continue;
		}

		// compute screen-space bounding rectangle
		float min_x = min(min(p0.x, p1.x), p2.x);
		float min_y = min(min(p0.y, p1.y), p2.y);
		float max_x = max(max(p0.x, p1.x), p2.x);
		float max_y = max(max(p0.y, p1.y), p2.y);

		// clamp to screen
		min_x = clamp(min_x, 0.0f, (float)target.width);
		min_y = clamp(min_y, 0.0f, (float)target.height);
		max_x = clamp(max_x, 0.0f, (float)target.width);
		max_y = clamp(max_y, 0.0f, (float)target.height);

		int size_x = ceil(max_x) - floor(min_x);
		int size_y = ceil(max_y) - floor(min_y);
		int numFragments = size_x * size_y;

		// iterate through fragments in bounding rectangle and draw if within triangle
		int numProcessedSamples = 0;
		for (int fragOffset = 0; fragOffset < numFragments; fragOffset += block.num_threads()) {

			// safety mechanism: don't draw more than <x> pixels per thread
			if (numProcessedSamples > 10'000)
				break;

			int fragID = fragOffset + block.thread_rank();
			int fragX = fragID % size_x;
			int fragY = fragID / size_x;

			float2 pFrag = {floor(min_x) + float(fragX), floor(min_y) + float(fragY)};
			float2 sample = {pFrag.x - p0.x, pFrag.y - p0.y};

			// v: vertex[0], s: vertex[1], t: vertex[2]
			float s = cross(sample, v02) / cross(v01, v02);
			float t = cross(v01, sample) / cross(v01, v02);
			float v = 1.0 - (s + t);

			int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
			int pixelID = pixelCoords.x + pixelCoords.y * target.width;
			pixelID = clamp(pixelID, 0, int(target.width * target.height) - 1);

			if (s >= 0.0f) {
				if (t >= 0.0f) {
					if (v >= 0.0f) {
						uint32_t color = computeColor(triangleIndex, triangles, material, texture, s, t, v);

						float depth = v * p0.w + s * p1.w + t * p2.w;
						uint64_t udepth = *((uint32_t*)&depth);
						uint64_t pixel = (udepth << 32ull) | color;

						atomicMin(&target.framebuffer[pixelID], pixel);
					}
				}
			}

			numProcessedSamples++;
		}
	}
}

template <typename VertexShader>
void rasterizeTrianglesInstanced(TriangleData triangles, TriangleMaterial material, HeightmapInstance* instances, CTexture texture, RenderTarget target, VertexShader&& vertexShader) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	if (grid.thread_rank() == 0) {
		numProcessedTriangles = 0;
		veryLargeTriangleCounter = 0;

		//printf("drawing %i triangles and %i instances\n", int(triangles.count), int(triangles.instances));
	}

	grid.sync();

	mat4 transform = target.proj * target.view * triangles.transform;

	constexpr int TRIANGLES_PER_SWEEP = 32;
	__shared__ float3 sh_positions[3 * TRIANGLES_PER_SWEEP];
	__shared__ float2 sh_uvs[3 * TRIANGLES_PER_SWEEP];
	__shared__ int sh_blockTriangleOffset;

	__shared__ struct {
		int numTriangles;
		int indices[TRIANGLES_PER_SWEEP];
	} largeTriangleSchedule;

	while (true) {
		block.sync();
		if (block.thread_rank() == 0) {
			sh_blockTriangleOffset = atomicAdd(&numProcessedTriangles, TRIANGLES_PER_SWEEP);
			largeTriangleSchedule.numTriangles = 0;
		}
		block.sync();

		int numTrianglesInBlock = min(int(triangles.count * triangles.instances) - sh_blockTriangleOffset, TRIANGLES_PER_SWEEP);

		if (numTrianglesInBlock <= 0)
			break;

		// load triangles into shared memory
		for (int i = block.thread_rank(); i < numTrianglesInBlock; i += block.size()) {
			int index = sh_blockTriangleOffset + i;
			int triangleIndex = index % triangles.count;
			int instanceIndex = index / triangles.count;
			sh_positions[3 * i + 0] = vertexShader(3 * triangleIndex + 0, instanceIndex);
			sh_positions[3 * i + 1] = vertexShader(3 * triangleIndex + 1, instanceIndex);
			sh_positions[3 * i + 2] = vertexShader(3 * triangleIndex + 2, instanceIndex);

			sh_uvs[3 * i + 0] = triangles.uv[3 * triangleIndex + 0];
			sh_uvs[3 * i + 1] = triangles.uv[3 * triangleIndex + 1];
			sh_uvs[3 * i + 2] = triangles.uv[3 * triangleIndex + 2];
		}

		block.sync();

		// draw triangles
		for (int i = block.thread_rank(); i < numTrianglesInBlock; i += block.size()) {
			int index = sh_blockTriangleOffset + i;
			int triangleIndex = index % triangles.count;
			int instanceIndex = index / triangles.count;

			float3 v_0 = sh_positions[3 * i + 0];
			float3 v_1 = sh_positions[3 * i + 1];
			float3 v_2 = sh_positions[3 * i + 2];

			float3 normal = normalize(cross(v_1 - v_0, v_2 - v_1));

			float4 p_0 = toScreenCoord(v_0, transform, target.width, target.height);
			float4 p_1 = toScreenCoord(v_1, transform, target.width, target.height);
			float4 p_2 = toScreenCoord(v_2, transform, target.width, target.height);

			if (p_0.w < 0.0f || p_1.w < 0.0f || p_2.w < 0.0f)
				continue;

			float2 v_01 = {p_1.x - p_0.x, p_1.y - p_0.y};
			float2 v_02 = {p_2.x - p_0.x, p_2.y - p_0.y};

			auto cross = [](float2 a, float2 b) {
				return a.x * b.y - a.y * b.x;
			};

			{	// backface culling
				float w = cross(v_01, v_02);
				if (w < 0.0)
					continue;
			}

			// compute screen-space bounding rectangle
			float min_x = min(min(p_0.x, p_1.x), p_2.x);
			float min_y = min(min(p_0.y, p_1.y), p_2.y);
			float max_x = max(max(p_0.x, p_1.x), p_2.x);
			float max_y = max(max(p_0.y, p_1.y), p_2.y);

			// clamp to screen
			min_x = clamp(min_x, 0.0f, (float)target.width);
			min_y = clamp(min_y, 0.0f, (float)target.height);
			max_x = clamp(max_x, 0.0f, (float)target.width);
			max_y = clamp(max_y, 0.0f, (float)target.height);

			int size_x = ceil(max_x) - floor(min_x);
			int size_y = ceil(max_y) - floor(min_y);
			int numFragments = size_x * size_y;

			if (numFragments > 40'000) {
				uint32_t index = atomicAdd(&veryLargeTriangleCounter, 1);
				veryLargeTriangleIndices[index] = index;
				continue;
			} else if (numFragments > 1024) {
				// TODO: schedule block-wise rasterization
				uint32_t index = atomicAdd(&largeTriangleSchedule.numTriangles, 1);
				largeTriangleSchedule.indices[index] = index;
				continue;
			}

			int numProcessedSamples = 0;
			for (int fragOffset = 0; fragOffset < numFragments; fragOffset += 1) {

				// safety mechanism: don't draw more than <x> pixels per thread
				if (numProcessedSamples > 2000)
					break;

				int fragID = fragOffset;	// + block.thread_rank();
				int fragX = fragID % size_x;
				int fragY = fragID / size_x;

				float2 pFrag = {floor(min_x) + float(fragX), floor(min_y) + float(fragY)};
				float2 sample = {pFrag.x - p_0.x, pFrag.y - p_0.y};

				// v: vertex[0], s: vertex[1], t: vertex[2]
				float s = cross(sample, v_02) / cross(v_01, v_02);
				float t = cross(v_01, sample) / cross(v_01, v_02);
				float v = 1.0f - (s + t);

				int2 pixelCoords = make_int2(pFrag.x, pFrag.y);
				int pixelID = pixelCoords.x + pixelCoords.y * target.width;
				pixelID = clamp(pixelID, 0, int(target.width * target.height) - 1);

				if (s >= 0.0f) {
					if (t >= 0.0f) {
						if (v >= 0.0f) {
							float diff = abs(dot(normal, normalize(float3{-1.0f, -1.0f, -1.0f})));
							uint32_t c = colorFromIndex(instances[instanceIndex].heightmapIndex);
							auto rgba = (uint8_t*)&c;
							rgba[0] = static_cast<uint8_t>((static_cast<float>(rgba[0]) * 255.0 * diff) / 255.0);
							rgba[1] = static_cast<uint8_t>((static_cast<float>(rgba[1]) * 255.0 * diff) / 255.0);
							rgba[2] = static_cast<uint8_t>((static_cast<float>(rgba[2]) * 255.0 * diff) / 255.0);
							uint32_t color = c;//computeColor(triangleIndex, triangles, material, texture, s, t, v);

							float depth = v * p_0.w + s * p_1.w + t * p_2.w;
							uint64_t udepth = *((uint32_t*)&depth);
							uint64_t pixel = (udepth << 32ull) | color;

							atomicMin(&target.framebuffer[pixelID], pixel);
						}
					}
				}

				++numProcessedSamples;
			}
		}

		block.sync();

		// do blockwise rasterization for triangles that were too large for thread-wise rasterization
		rasterizeLargeTrianglesInstanced(
				triangles,
				material,
				texture,
				target,
				largeTriangleSchedule.indices,
				largeTriangleSchedule.numTriangles,
				transform,
				vertexShader);
	}

	grid.sync();
}

extern "C" __global__ void
kernel_drawTrianglesInstanced(TriangleData triangles, TriangleMaterial material, HeightmapInstance* instances, float* heightMaps, CTexture texture, RenderTarget target) {
	auto block = cg::this_thread_block();
	material.mode = MATERIAL_MODE_UVS;
	rasterizeTrianglesInstanced(triangles, material, instances, texture, target, [&](const int vertexIndex, const int instanceIndex) {
		auto instance = instances[instanceIndex];
		size_t heightmapSize = 64;
		size_t heightmapOffset = instance.heightmapIndex * heightmapSize * heightmapSize;

		float metersPerPixel = 1.0f;//10.0f;
		float contextRadius = 1.5f;
		float numericalStabilityFactor = 10.0f;

		float patchRadius = sqrt(2.0f) * 0.5f * static_cast<float>(heightmapSize) * contextRadius * metersPerPixel;

		float3 vertex = triangles.position[vertexIndex];
		float2 uv = triangles.uv[vertexIndex];

		uint2 heightmapUv = uint2{
				static_cast<uint32_t>(floor(uv.x * static_cast<float>(heightmapSize))),
				static_cast<uint32_t>(floor(uv.y * static_cast<float>(heightmapSize)))
		};

		vertex.x += instance.offsetX;
		vertex.y += instance.offsetY;
		vertex.z = heightMaps[heightmapOffset + heightmapUv.x * heightmapSize + heightmapUv.y] * patchRadius / numericalStabilityFactor;

		return vertex;
	});
}
