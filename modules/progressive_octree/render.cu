#include "hip/hip_runtime.h"
// Some code in this file, particularly frustum, ray and intersection tests,
// is adapted from three.js. Three.js is licensed under the MIT license
// This file this follows the three.js licensing
// License: MIT https://github.com/mrdoob/three.js/blob/dev/LICENSE

#define CUB_DISABLE_BF16_SUPPORT

#define GLM_FORCE_CUDA
#define GLM_ENABLE_EXPERIMENTAL
#define CUDA_VERSION 12000

namespace std{
	using size_t = ::size_t;
};

using namespace std;

#include "./libs/glm/glm/glm.hpp"

#include <hip/hip_cooperative_groups.h>
#include <hiprand/hiprand_kernel.h>

#include "HostDeviceInterface.h"
#include "builtin_types.h"
#include "hip/hip_vector_types.h"
#include "utils.cuh"

#include "triangles.cuh"

#include "math.cuh"
#include "rasterization.cuh"


namespace cg = cooperative_groups;

constexpr int SPLAT_SIZE = 1;
constexpr uint32_t BACKGROUND_COLOR = 0x00332211;

// https://colorbrewer2.org/
uint32_t SPECTRAL[8] = {
		0x4f3ed5,
		0x436df4,
		0x61aefd,
		0x8be0fe,
		0x98f5e6,
		0xa4ddab,
		0xa5c266,
		0xbd8832,
};

// struct PointBatch {
// 	uint32_t numPoints;
// 	Point points[50'000];
// };

// template <typename T>
// struct Pool {

// 	T** items;
// 	int32_t numItems;
// };

// struct Data {
// 	uint32_t* buffer;
// 	// Pool<PointBatch>* pointBatchPool;
// 	uint64_t* framebuffer;
// 	// Lines* lines;
// 	// Triangles* triangles;
// 	// uint32_t* chunkInitSumColors;
// };

struct SampleInfo{
	vec2 uv_global;
	uint32_t texels_x;
	uint32_t texels_y;
	uint32_t texel_x_global;
	uint32_t texel_y_global;
	uint32_t patch_x;
	uint32_t patch_y;
	uint32_t patchID;
	uint32_t numPatches;
};

SampleInfo worldPosToSampleInfo(vec3 worldPos, uint32_t patches_x, uint32_t patches_y, uint32_t heightmapSize){
	vec3 boxSize = {
		640.0 * float(patches_x), 
		640.0 * float(patches_y),
		0.0, 
	};
	vec2 uv_global = {
		(worldPos.x + 5.0f) / boxSize.x,
		(worldPos.y + 5.0f) / boxSize.y,
	};

	uint32_t texels_x = patches_x * heightmapSize;
	uint32_t texels_y = patches_y * heightmapSize;

	int numPatches = patches_x * patches_y;

	uint32_t texel_x_global = uv_global.x * texels_x;
	uint32_t texel_y_global = uv_global.y * texels_y;

	int patch_x = texel_x_global / 64.0;
	int patch_y = texel_y_global / 64.0;
	int patchID = patch_x + patches_x * patch_y;

	SampleInfo info;
	info.uv_global = uv_global;
	info.texel_x_global = texel_x_global;
	info.texel_y_global = texel_y_global;
	info.texels_x = texels_x;
	info.texels_y = texels_y;
	info.patch_x = patch_x;
	info.patch_y = patch_y;
	info.patchID = patchID;
	info.numPatches = numPatches;

	return info;
}

void toScreen(vec3 boxMin, vec3 boxMax, vec2& screen_min, vec2& screen_max, mat4 transform, float width, float height) {
	auto min8 = [](float f0, float f1, float f2, float f3, float f4, float f5, float f6, float f7) {
		float m0 = min(f0, f1);
		float m1 = min(f2, f3);
		float m2 = min(f4, f5);
		float m3 = min(f6, f7);

		float n0 = min(m0, m1);
		float n1 = min(m2, m3);

		return min(n0, n1);
	};

	auto max8 = [](float f0, float f1, float f2, float f3, float f4, float f5, float f6, float f7) {
		float m0 = max(f0, f1);
		float m1 = max(f2, f3);
		float m2 = max(f4, f5);
		float m3 = max(f6, f7);

		float n0 = max(m0, m1);
		float n1 = max(m2, m3);

		return max(n0, n1);
	};

	// compute node boundaries in screen space
	vec4 p000 = {boxMin.x, boxMin.y, boxMin.z, 1.0f};
	vec4 p001 = {boxMin.x, boxMin.y, boxMax.z, 1.0f};
	vec4 p010 = {boxMin.x, boxMax.y, boxMin.z, 1.0f};
	vec4 p011 = {boxMin.x, boxMax.y, boxMax.z, 1.0f};
	vec4 p100 = {boxMax.x, boxMin.y, boxMin.z, 1.0f};
	vec4 p101 = {boxMax.x, boxMin.y, boxMax.z, 1.0f};
	vec4 p110 = {boxMax.x, boxMax.y, boxMin.z, 1.0f};
	vec4 p111 = {boxMax.x, boxMax.y, boxMax.z, 1.0f};

	vec4 ndc000 = transform * p000;
	vec4 ndc001 = transform * p001;
	vec4 ndc010 = transform * p010;
	vec4 ndc011 = transform * p011;
	vec4 ndc100 = transform * p100;
	vec4 ndc101 = transform * p101;
	vec4 ndc110 = transform * p110;
	vec4 ndc111 = transform * p111;

	vec4 s000 = ((ndc000 / ndc000.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s001 = ((ndc001 / ndc001.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s010 = ((ndc010 / ndc010.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s011 = ((ndc011 / ndc011.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s100 = ((ndc100 / ndc100.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s101 = ((ndc101 / ndc101.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s110 = ((ndc110 / ndc110.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};
	vec4 s111 = ((ndc111 / ndc111.w) * 0.5f + 0.5f) * vec4{width, height, 1.0f, 1.0f};

	float smin_x = min8(s000.x, s001.x, s010.x, s011.x, s100.x, s101.x, s110.x, s111.x);
	float smin_y = min8(s000.y, s001.y, s010.y, s011.y, s100.y, s101.y, s110.y, s111.y);

	float smax_x = max8(s000.x, s001.x, s010.x, s011.x, s100.x, s101.x, s110.x, s111.x);
	float smax_y = max8(s000.y, s001.y, s010.y, s011.y, s100.y, s101.y, s110.y, s111.y);

	screen_min.x = smin_x;
	screen_min.y = smin_y;
	screen_max.x = smax_x;
	screen_max.y = smax_y;
}

// from: https://stackoverflow.com/a/51549250
// TODO: License
__forceinline__ float atomicMinFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMin((int*)addr, __float_as_int(value)))
										 : __uint_as_float(atomicMax((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

// from: https://stackoverflow.com/a/51549250
// TODO: License
__forceinline__ float atomicMaxFloat(float* addr, float value) {
	float old;
	old = (value >= 0) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value)))
										 : __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));

	return old;
}

extern "C" __global__ void kernel_chunkLoaded(
		const uint32_t chunkIndex,
		const uint32_t chunkID,
		Point* ptr_points,
		Tile* tiles,
		Chunk* chunks,
		uint32_t* chunkInitSumColors,
		SparseHeightmapPointer* sparseHeightmapPointers,
		uint32_t patches_x, uint32_t patches_y,
		uint32_t* accumulate,
		const Patches patches
){

	auto grid = cg::this_grid();

	// if(grid.thread_rank() == 0){
	// 	printf("chunk loaded\n");
	// }

	Chunk& chunk = chunks[chunkID];
	chunk.state = STATE_LOADED;

	Tile& tile = tiles[chunk.tileID];

	chunk.points = ptr_points;

	// compute new bounding box and sum of colors
	grid.sync();
	if (grid.thread_rank() == 0) {
		chunkInitSumColors[0] = 0;
		chunkInitSumColors[1] = 0;
		chunkInitSumColors[2] = 0;
		chunkInitSumColors[3] = 0;
	}

	grid.sync();

	processRange(chunk.numPoints, [&](int index) {
		Point point = chunk.points[index];
		atomicAdd(&chunkInitSumColors[0], uint32_t(point.rgba[0]));
		atomicAdd(&chunkInitSumColors[1], uint32_t(point.rgba[1]));
		atomicAdd(&chunkInitSumColors[2], uint32_t(point.rgba[2]));
		atomicAdd(&chunkInitSumColors[3], 1u);
	});

	grid.sync();

	// set chunk's color to average of point
	if (grid.thread_rank() == 0) {
		chunk.rgba[0] = chunkInitSumColors[0] / chunkInitSumColors[3];
		chunk.rgba[1] = chunkInitSumColors[1] / chunkInitSumColors[3];
		chunk.rgba[2] = chunkInitSumColors[2] / chunkInitSumColors[3];
	}

	int heightmapSize = 64;
	SampleInfo sample_min = worldPosToSampleInfo(tile.min, patches_x, patches_y, heightmapSize);
	SampleInfo sample_max = worldPosToSampleInfo(tile.max, patches_x, patches_y, heightmapSize);

	int numPatchesX = (sample_max.patch_x - sample_min.patch_x) + 1;
	int numPatchesY = (sample_max.patch_y - sample_min.patch_y) + 1;
	int start_x = sample_min.patch_x;
	int end_x = start_x + min(numPatchesX, 2);
	int start_y = sample_min.patch_y;
	int end_y = start_y + min(numPatchesY, 2);

	for(int x = start_x; x < end_x; x++)
	for(int y = start_y; y < end_y; y++)
	{
		
		int patchID = x + patches_x * y;
		SparseHeightmapPointer ptr = sparseHeightmapPointers[patchID];
		Patch& patch = patches.patches[ptr.patchIndex];

		if(!patch.hasHeightmap) continue;

		if(grid.thread_rank() < 64 * 64){
			accumulate[4 * grid.thread_rank() + 0] = 0;
			accumulate[4 * grid.thread_rank() + 1] = 0;
			accumulate[4 * grid.thread_rank() + 2] = 0;
			accumulate[4 * grid.thread_rank() + 3] = 0;
		}

		grid.sync();

		processRange(chunk.numPoints, [&](int index){
			Point point = chunk.points[index];

			SampleInfo sample = worldPosToSampleInfo({point.x, point.y, point.z}, patches_x, patches_y, heightmapSize);

			if(sample.patch_x != x || sample.patch_y != y) return;

			int tx = sample.texel_x_global % 64;
			int ty = sample.texel_y_global % 64;
			int texelID = tx + 64 * ty;

			if(texelID >= 0 && texelID < 64 * 64){
				
				atomicAdd(&accumulate[4 * texelID + 0], point.rgba[0]);
				atomicAdd(&accumulate[4 * texelID + 1], point.rgba[1]);
				atomicAdd(&accumulate[4 * texelID + 2], point.rgba[2]);
				atomicAdd(&accumulate[4 * texelID + 3], 1);
			}
		});

		grid.sync();

		if(grid.thread_rank() < 64 * 64){
			uint32_t texelID = grid.thread_rank();
			uint32_t R = accumulate[4 * texelID + 0];
			uint32_t G = accumulate[4 * texelID + 1];
			uint32_t B = accumulate[4 * texelID + 2];
			uint32_t A = accumulate[4 * texelID + 3];

			if(A > 2){
				uint32_t color = 0;
				uint8_t* rgba = (uint8_t*)&color;
				rgba[0] = R / A;
				rgba[1] = G / A;
				rgba[2] = B / A;
				// rgba[3] = 255;
				rgba[3] = (patch.texture[texelID] >> 24);

				patch.texture[texelID] = color;
			}
		}


	}

	
	
	// tile.min


	// // update textures
	// processRange(chunk.numPoints, [&](int index) {

	// });

	// SampleInfo sample = worldPosToSampleInfo({point.x, point.y, point.z}, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);
	// SparseHeightmapPointer ptr = sparseHeightmapPointers[sample.patchID];
	// Patch& patch = patches.patches[ptr.patchIndex];

	// if(ptr.heightmap && chunk.hasUpdatedTexture == 0 && chunk.numPoints == 50'000){
	// 	uint32_t tx = sample.texel_x_global % 64;
	// 	uint32_t ty = sample.texel_y_global % 64;
	// 	uint32_t texelID = tx + 64 * ty;
	// 	// patch.texture[texelID] = point.color;
	// 	// atomicMin(&patch.texture[texelID], point.color);

	// 	uint64_t c64;
	// 	uint16_t* rgba64 = (uint16_t*)&c64;
	// 	uint32_t color = point.color;
	// 	uint8_t* rgba = (uint8_t*)&color;

	// 	rgba64[0] = rgba[0];
	// 	rgba64[1] = rgba[1];
	// 	rgba64[2] = rgba[2];
	// 	rgba64[3] = 1;
	// 	patch.texture[texelID] = c64;

	// 	// uint64_t old = atomicAdd(&patch.texture[texelID], c64);


	// 	// atomicMin(&patch.texture[texelID], c64);
	// }
}

extern "C" __global__ void kernel_chunkUnloaded(const uint32_t chunkID, Chunk* chunks) {
	Chunk& chunk = chunks[chunkID];
	chunk.state = STATE_EMPTY;
	chunk.points = nullptr;
}

uint32_t colorFromIndex(uint32_t index) {
	return ((index % 127) + 1) * 12345678;
}


extern "C" __global__ 
void kernel_clearFramebuffer(uint64_t* framebuffer, uint32_t numPixels){
		processRange(0, numPixels, [&](int pixelIndex) {
		// depth:						7f800000 (Infinity)
		// background color: 00332211 (aabbggrr)
		// framebuffer[pixelIndex] = 0x7f800000'00332211ull;
		framebuffer[pixelIndex] = (0x7f800000ull << 32) | uint64_t(BACKGROUND_COLOR);
	});
}

extern "C" __global__ 
void kernel_check_visibility(
		const Uniforms uniforms,
		RenderTarget renderTarget,
		hipSurfaceObject_t gl_colorbuffer,
		TriangleData trianglesData,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		Commands commands,
		uint32_t* patchesPoints,
		uint32_t* patchesTriangles
) {
	auto tStart = nanotime();

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	// todo: either use extra buffers for kernel specific allocations or move all of them into data to avoid issues with overlapping executions of other kernels that do the same thing
	//	-> I actually ran into this issue with kernel_render & kernel_chunkLoaded - moved kernel_chunkLoaded extra allocations into data for now

	grid.sync();

	if (grid.thread_rank() == 0) {
		*commands.numChunksToLoad       = 0;
		state->patchAsPointsQueue       = 0;
		state->patchAsTrianglesQueue    = 0;
		state->patchCounter             = 0;
		state->numHighlyVisibleLasTiles = 0;
		state->numChunksVisible         = 0;
		state->numPointsInChunksVisible = 0;
	}

	grid.sync();

	// CHECK VISIBILITY OF LAS TILES
	if (grid.thread_rank() < uniforms.numTiles) {
		Tile& tile = lasTiles.tiles[grid.thread_rank()];


		vec3 offset = {0.0f, 0.0f, 0.0f};
		bool isInFrustum = intersectsFrustum(uniforms.transform_updateBound, 
			tile.min + offset, 
			tile.max + offset
		);
		// isInFrustum = false;

		// bool isInsideX = uniforms.cameraPosition.x > tile.min.x && uniforms.cameraPosition.x < tile.max.x;
		// bool isInsideY = uniforms.cameraPosition.y > tile.min.y && uniforms.cameraPosition.y < tile.max.y;
		// bool isInsideZ = uniforms.cameraPosition.z > tile.min.z && uniforms.cameraPosition.z < tile.max.z;
		// bool isInAABB = isInsideX && isInsideY;

		if (uniforms.disableHighResTiles) {
			tile.isHighlyVisible = false;
		} 
		else if (isInFrustum) 
		{
			// vec3 pos = vec3{tile.min.x + tile.max.x, tile.min.y + tile.max.y, tile.min.z + tile.max.z} * 0.5f;
			// vec3 size = vec3{tile.max.x - tile.min.x, tile.max.y - tile.min.y, tile.max.z - tile.min.z} * 0.5f;

			vec3 center = (tile.min + tile.max) * 0.5f;
			vec3 size = tile.max - tile.min;

			vec4 ndcCenter = uniforms.transform_updateBound * vec4(center, 1.0f);
			vec2 screenCenter = vec2{ndcCenter.x, ndcCenter.y} / ndcCenter.w;
			float screenDist = clamp(1.0f - glm::length(screenCenter), 0.0f, 1.0f);

			float w_center = screenDist;// / ndcCenter.w;

			// if(ndcCenter.w < 0.0f){
			// 	w = 0.0f;
			// }

			vec2 smin, smax;
			toScreen(tile.min, tile.max, smin, smax, uniforms.transform_updateBound, uniforms.width, uniforms.height);

			// screen-space size
			float dx = smax.x - smin.x;
			float dy = smax.y - smin.y;

			float screen_center_x = ((smin.x + smax.x) * 0.5f - uniforms.width * 0.5f) / uniforms.width;
			float screen_center_y = ((smin.y + smax.y) * 0.5f - uniforms.height * 0.5f) / uniforms.height;
			float d = sqrt(screen_center_x * screen_center_x + screen_center_y * screen_center_y);

			float w = clamp(exp(-d * d / 0.040f), 0.1f, 1.0f) * dx * dy;
			w = w * w_center;

			// normalize by number of points. 
			// Smaller tiles with fewer points should also get a good chance to be visble.
			float w_count = float(tile.numPoints) / 1'000'000.0f;
			w_count = clamp(w_count, 0.1f, 10.0f);
			w = w / w_count;


			// bool isHighlyVisible = w > 0.00025f;
			bool isHighlyVisible = w > 10000.0f;
			// tile.isHighlyVisible = isHighlyVisible;
			// bool isHighlyVisible = false;
			tile.weight = w;
			tile.isHighlyVisible = isHighlyVisible;

			if(isHighlyVisible){
				uint32_t index = atomicAdd(&state->numHighlyVisibleLasTiles, 1);
				state->highlyVisibleLasTileIds[index] = grid.thread_rank();
			}

			// DEBUG: safeguard, limit max number of tiles
			// if(isHighlyVisible){
			// 	uint32_t index = atomicAdd(&state->numHighlyVisibleLasTiles, 1);

			// 	if(index < 40){
			// 		tile.isHighlyVisible = true;
			// 	}
			// }

			// tile.isHighlyVisible = false;

			// if (tile.isHighlyVisible) {
			// 	uint32_t index = atomicAdd(&state->numHighlyVisibleLasTiles, 1);

			// 	// if(index > 1){
			// 	// 	atomicSub(&state->numHighlyVisibleLasTiles, 1);
			// 	// }
			// 	// else
			// 	// {
			// 		state->highlyVisibleLasTileIds[index] = grid.thread_rank();
			// 	// }

				
			// 	// printf("[device] highly visible! tileID: %d, state: %d\n", grid.thread_rank(), tile.state);
			// }
		}else if(!isInFrustum){
			tile.isHighlyVisible = false;
		}
	}

	grid.sync();

	if(grid.thread_rank() == 0){
		// printf("state->numHighlyVisibleLasTiles: %d\n", state->numHighlyVisibleLasTiles);
		// printf("uniforms.numChunks: %d\n", uniforms.numChunks);
		// printf("uniforms.numTiles: %d\n", uniforms.numTiles);
	}

	// HANDLE CHUNKS (corresponds to heightmaps)
	processRange(uniforms.numChunks, [&](int chunkIndex) {
		Chunk& chunk = lasTiles.chunks[chunkIndex];
		Tile& tile = lasTiles.tiles[chunk.tileID];

		chunk.weight = tile.weight;

		// if(tile.isHighlyVisible){
		// 	printf("chunkIndex %d \n",chunkIndex);
		// }

		// CHECK IF NEEDS TO BE UNLOADED
		if (!tile.isHighlyVisible && !intersectsFrustum(uniforms.transform_updateBound, chunk.min, chunk.max)) {

			if (chunk.state == STATE_LOADED) {
				Command command;
				command.command = CMD_UNLOAD_CHUNK;

				CommandUnloadChunkData cmddata;
				cmddata.tileID = chunk.tileID;
				cmddata.chunkIndex = chunk.chunkIndex;
				cmddata.chunkID = chunkIndex;
				cmddata.cptr_pointBatch = (uint64_t)chunk.points;

				memcpy(command.data, &cmddata, sizeof(cmddata));

				uint32_t index = atomicAdd(commands.commandQueueCounter, 1llu) % COMMAND_QUEUE_CAPACITY;
				commands.commandQueue[index] = command;

				chunk.state = STATE_UNLOADING;
			}
			return;
		}

		vec2 smin, smax;
		toScreen(chunk.min, chunk.max, smin, smax, uniforms.transform_updateBound, uniforms.width, uniforms.height);

		// screen-space size
		float dx = smax.x - smin.x;
		float dy = smax.y - smin.y;

		float screen_center_x = ((smin.x + smax.x) * 0.5f - uniforms.width * 0.5f) / uniforms.width;
		float screen_center_y = ((smin.y + smax.y) * 0.5f - uniforms.height * 0.5f) / uniforms.height;
		float d = sqrt(screen_center_x * screen_center_x + screen_center_y * screen_center_y);

		vec3 pos = vec3{chunk.min.x + chunk.max.x, chunk.min.y + chunk.max.y, chunk.min.z + chunk.max.z} * 0.5f;
		vec3 size = vec3{chunk.max.x - chunk.min.x, chunk.max.y - chunk.min.y, chunk.max.z - chunk.min.z} * 0.5f;

		if (chunk.state == STATE_LOADED && chunk.chunkIndex % 1 == 0) {
			uint32_t boxColor = 0x0000ff00;
		}


		if (tile.isHighlyVisible) {
			// LOAD CHUNK, IF NEEDED
			if (chunk.state != STATE_LOADED) {
				uint32_t index_nctl = atomicAdd(commands.numChunksToLoad, 1);

				if (index_nctl < MAX_CHUNKS_TO_LOAD) {
					commands.chunksToLoad[index_nctl] = chunkIndex;
				}
			}

			// printf("highly visible. %d\n", chunk.tileID);

			if (chunk.state == STATE_EMPTY) {
				// LOAD CHUNK!
				// printf("[device] load chunk! tileID: %d, chunkID: %d \n", chunk.tileID, chunkIndex);
				Command command;
				command.command = CMD_READ_CHUNK;

				CommandReadChunkData cmddata;
				cmddata.tileID = chunk.tileID;
				cmddata.chunkIndex = chunk.chunkIndex;
				cmddata.chunkID = chunkIndex;
				cmddata.cptr_pointBatch = 0;

				memcpy(command.data, &cmddata, sizeof(cmddata));

				uint32_t index = atomicAdd(commands.commandQueueCounter, 1llu) % COMMAND_QUEUE_CAPACITY;
				commands.commandQueue[index] = command;

				chunk.state = STATE_LOADING;
			}
		} else if (chunk.state == STATE_LOADED && !tile.isHighlyVisible) {
			// chunk is loaded but not high priority -> unload points
			Command command;
			command.command = CMD_UNLOAD_CHUNK;

			CommandUnloadChunkData cmddata;
			cmddata.tileID = chunk.tileID;
			cmddata.chunkIndex = chunk.chunkIndex;
			cmddata.chunkID = chunkIndex;
			cmddata.cptr_pointBatch = (uint64_t)chunk.points;

			memcpy(command.data, &cmddata, sizeof(cmddata));

			uint32_t index = atomicAdd(commands.commandQueueCounter, 1llu) % COMMAND_QUEUE_CAPACITY;
			commands.commandQueue[index] = command;

			chunk.state = STATE_UNLOADING;

			// drawBoundingBox(data->lines, pos, 2.02f * size, 0x000000ff);
		}
	});

	grid.sync();

	// CHECK WHICH PATCHES TO RENDER AS POINTS OR TRIANGLES
	processRange(patches.numPatches, [&](const size_t i) {
		Patch& patch = patches.patches[i];

		patch.isVisible = false;

		// check if visible
		if (!intersectsFrustum(uniforms.transform_updateBound, patch.min - 150.0f, patch.max + 150.0f)) {
			return;
		}

		// todo: the rest of this function could / should be in an extra pass

		// check if fully enclosed by highly visible las tile or if there are overlaps (then we need to check their fragments)
		// patch.noOverlaps = true;
		// for (size_t i = 0; i < state->numHighlyVisibleLasTiles; ++i) {
		// 	const auto& lasTile = lasTiles.tiles[state->highlyVisibleLasTileIds[i]];
		// 	if (!(patch.min.x < lasTile.min.x || patch.max.x > lasTile.max.x || patch.min.y < lasTile.min.y || patch.max.y > lasTile.max.y)) {
		// 		return;
		// 	}
		// 	patch.noOverlaps &= patch.max.x < lasTile.min.x || lasTile.max.x < patch.min.x || patch.max.y < lasTile.min.y || lasTile.max.y < patch.min.y;
		// }

		patch.isVisible = true;

		if (uniforms.forceChunkPointsAndHeightmaps) {
			if (patch.hasHeightmap) 
			{
				uint32_t index = atomicAdd(&state->patchAsTrianglesQueue, 1);
				patchesTriangles[index] = i;
			}
			{
				uint32_t index = atomicAdd(&state->patchAsPointsQueue, 1);
				patchesPoints[index] = i;
			}
		} else {
			if (patch.hasHeightmap && !uniforms.disableHeightmaps) {
				uint32_t index = atomicAdd(&state->patchAsTrianglesQueue, 1);
				patchesTriangles[index] = i;
			} else if (!uniforms.disableChunkPoints) {
				uint32_t index = atomicAdd(&state->patchAsPointsQueue, 1);
				patchesPoints[index] = i;
			}
		}
	});

	grid.sync();

	processRange(MAX_CHUNKS_TO_LOAD, [&](int index) {
		if (index >= *commands.numChunksToLoad) {
			commands.chunksToLoad[index] = -1;
		}
	});

}

extern "C" __global__ 
void kernel_update_numVisibleChunks(
		Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		Chunk* visibleChunks
) {
	uint32_t chunkIndex = cg::this_grid().thread_rank();

	if(chunkIndex >= uniforms.numChunks) return;

	Chunk& chunk = lasTiles.chunks[chunkIndex];

	if (chunk.state == STATE_LOADED) {
		uint32_t index = atomicAdd(&state->numChunksVisible, 1);
		atomicAdd(&state->numPointsInChunksVisible, chunk.numPoints);
		visibleChunks[index] = chunk;
	}
}

extern "C" __global__ 
void kernel_render_visibleChunks_FullressPoints_depth(
		Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		Chunk* visibleChunks,
		float* depthbuffer,
		uint32_t* colorbuffer
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uint32_t chunkIndex = grid.block_rank();

	// if(chunkIndex == 1810 && block.thread_rank() == 0){
	// 	printf("state->numChunksVisible: %d \n", state->numChunksVisible);
	// }

	if(chunkIndex >= state->numChunksVisible) return;

	Chunk& chunk = visibleChunks[chunkIndex];

	// if(chunk.state != STATE_LOADED) return;

	for(
		int pointIndex = block.thread_rank();
		pointIndex < chunk.numPoints;
		pointIndex += block.num_threads()
	){
		Point point = chunk.points[pointIndex];
		vec4 ndc = uniforms.transform * vec4{point.x, point.y, point.z, 1.0f};

		ndc.x = ndc.x / ndc.w;
		ndc.y = ndc.y / ndc.w;
		ndc.z = ndc.z / ndc.w;
		float depth = ndc.w;

		if(depth < 0.0f) continue;

		int x = (ndc.x * 0.5 + 0.5) * uniforms.width;
		int y = (ndc.y * 0.5 + 0.5) * uniforms.height;

		if (x > 1 && x < uniforms.width - 2.0) 
		if (y > 1 && y < uniforms.height - 2.0) 
		{

			uint32_t pixelID = x + int(uniforms.width) * y;
			uint32_t udepth = __float_as_uint(depth);

			if(depth < depthbuffer[pixelID]){
				atomicMin(&((uint32_t*)depthbuffer)[pixelID], udepth);
			}
		}
	
	}
}

extern "C" __global__ 
void kernel_render_visibleChunks_FullressPoints_color(
		Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		Chunk* visibleChunks,
		float* depthbuffer,
		uint32_t* colorbuffer
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uint32_t chunkIndex = grid.block_rank();

	if(chunkIndex >= state->numChunksVisible) return;

	Chunk& chunk = visibleChunks[chunkIndex];

	for(
		int pointIndex = block.thread_rank();
		pointIndex < chunk.numPoints;
		pointIndex += block.num_threads()
	){
		Point point = chunk.points[pointIndex];
		vec4 ndc = uniforms.transform * vec4{point.x, point.y, point.z, 1.0f};

		ndc.x = ndc.x / ndc.w;
		ndc.y = ndc.y / ndc.w;
		ndc.z = ndc.z / ndc.w;
		float depth = ndc.w;

		if(depth < 0.0f) continue;

		int x = (ndc.x * 0.5 + 0.5) * uniforms.width;
		int y = (ndc.y * 0.5 + 0.5) * uniforms.height;

		if (x > 1 && x < uniforms.width - 2.0) 
		if (y > 1 && y < uniforms.height - 2.0) 
		{
			uint32_t pixelID = x + int(uniforms.width) * y;
			
			float olddepth = depthbuffer[pixelID];

			if(depth < olddepth * 1.01f){
				atomicAdd(&colorbuffer[4 * pixelID + 0], point.rgba[0]);
				atomicAdd(&colorbuffer[4 * pixelID + 1], point.rgba[1]);
				atomicAdd(&colorbuffer[4 * pixelID + 2], point.rgba[2]);
				atomicAdd(&colorbuffer[4 * pixelID + 3], 1);
			}

			
		}
	
	}
}

extern "C" __global__ 
void kernel_render_visibleChunks_FullressPoints_resolve(
		Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		Chunk* visibleChunks,
		float* depthbuffer,
		uint32_t* colorbuffer
) {
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	uint32_t pixelID = grid.thread_rank();

	// if(pixelID > 1000000) return;

	uint32_t numPixels = renderTarget.width * renderTarget.height;
	if(pixelID >= numPixels) return;

	uint64_t color = 0;
	uint8_t* rgba = (uint8_t*)&color;

	float depth = depthbuffer[pixelID];
	uint32_t R = colorbuffer[4 * pixelID + 0];
	uint32_t G = colorbuffer[4 * pixelID + 1];
	uint32_t B = colorbuffer[4 * pixelID + 2];
	uint32_t count = colorbuffer[4 * pixelID + 3];
	rgba[0] = R / count;
	rgba[1] = G / count;
	rgba[2] = B / count;

	// if(depth > 0.0f && depth != Infinity){
	// 	color = 0xff00ff00;
	// }

	if(count == 0) return;

	uint64_t udepth = __float_as_uint(depth);
	// udepth = (0x7f800000llu) << 32;
	uint64_t encoded = (udepth << 32) | color;

	renderTarget.framebuffer[pixelID] = encoded;

}

// __device__ Chunk g_visibleChunks[1'000'000];
__device__ uint32_t g_numVisibleChunksProcessed;

extern "C" __global__ 
void kernel_render_tile_fullress_points(
		Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		SparseHeightmapPointer* sparseHeightmapPointers,
		LasTiles lasTiles,
		Chunk* visibleChunks
) {

	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	grid.sync();

	processRange(uniforms.numChunks, [&](int chunkIndex) {
		Chunk& chunk = lasTiles.chunks[chunkIndex];

		if (chunk.state == STATE_LOADED) {
			uint32_t index = atomicAdd(&state->numChunksVisible, 1);
			atomicAdd(&state->numPointsInChunksVisible, chunk.numPoints);
			visibleChunks[index] = chunk;
		}
	});

	grid.sync();

	if(grid.thread_rank() == 0){
		g_numVisibleChunksProcessed = 0;
	}
	grid.sync();

	__shared__ int sh_chunkIndex;

	while(true){
		block.sync();

		if(block.thread_rank() == 0){
			sh_chunkIndex = atomicAdd(&g_numVisibleChunksProcessed, 1);
		}

		block.sync();

		if(sh_chunkIndex >= state->numChunksVisible) break;

		Chunk& chunk = visibleChunks[sh_chunkIndex];

		for(
			int pointIndex = block.thread_rank();
			pointIndex < chunk.numPoints;
			pointIndex += block.num_threads()
		){
			Point point = chunk.points[pointIndex];

			// uint32_t color = 0xff000000;
			// uint8_t* rgba = (uint8_t*)&color;
			// rgba[0] = clamp(chunk.weight / 500.0f, 0.0f, 255.0f);

			// point.color = color;

			// SampleInfo sample = worldPosToSampleInfo({point.x, point.y, point.z}, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);
			// SparseHeightmapPointer ptr = sparseHeightmapPointers[sample.patchID];
			// Patch& patch = patches.patches[ptr.patchIndex];

			// if(ptr.heightmap && chunk.hasUpdatedTexture == 0 && chunk.numPoints == 50'000){
			// 	uint32_t tx = sample.texel_x_global % 64;
			// 	uint32_t ty = sample.texel_y_global % 64;
			// 	uint32_t texelID = tx + 64 * ty;
			// 	// patch.texture[texelID] = point.color;
			// 	// atomicMin(&patch.texture[texelID], point.color);

			// 	uint64_t c64;
			// 	uint16_t* rgba64 = (uint16_t*)&c64;
			// 	uint32_t color = point.color;
			// 	uint8_t* rgba = (uint8_t*)&color;

			// 	rgba64[0] = rgba[0];
			// 	rgba64[1] = rgba[1];
			// 	rgba64[2] = rgba[2];
			// 	rgba64[3] = 1;
			// 	patch.texture[texelID] = c64;

			// 	// uint64_t old = atomicAdd(&patch.texture[texelID], c64);


			// 	// atomicMin(&patch.texture[texelID], c64);
			// }

			// point.color = ptr.patchIndex * 123456;
			// point.color = chunk.tileID * 123456;
			
			rasterizePoint(
				point, renderTarget.framebuffer, uniforms.width, uniforms.height, 
				uniforms.transform, uniforms.pointSize);
		}

		// block.sync();

		// chunk.hasUpdatedTexture = 1;

	}
}


uint32_t sampleColor_nearest_sparse(
	SparseHeightmapPointer* sparseHeightmapPointers,
	uint32_t texel_x_global, uint32_t texel_y_global,
	uint32_t patches_x, uint32_t patches_y, uint32_t heightmapSize,
	const Patches patches
){
	int numPatches = patches_x * patches_y;
	
	uint32_t texel_x = texel_x_global % heightmapSize;
	uint32_t texel_y = texel_y_global % heightmapSize;
 
	int texelID = int(texel_x) * heightmapSize + int(texel_y);

	int patch_x = texel_x_global / 64.0;
	int patch_y = texel_y_global / 64.0;
	int patchID = patch_x + patches_x * patch_y;

	if(patchID >= 0 && patchID < numPatches){
		SparseHeightmapPointer ptr = sparseHeightmapPointers[patchID];

		int texelID = int(texel_x) + heightmapSize * int(texel_y);

		Patch& patch = patches.patches[ptr.patchIndex];

		if(patch.hasHeightmap){
			if(texelID < 64 * 64){
				uint32_t color = patch.texture[texelID];
				return color;
			}else{
				return 0xffff00ff;
			}
		}else{
			return 0;
		}
	}

	return 0x00000000;
}

uint32_t sampleColor_linear_sparse(
	SparseHeightmapPointer* sparseHeightmapPointers,
	vec2 uv_global,
	uint32_t patches_x, uint32_t patches_y, uint32_t heightmapSize,
	const Patches patches
){
	int numPatches = patches_x * patches_y;
	
	// uint32_t texel_x = texel_x_global % heightmapSize;
	// uint32_t texel_y = texel_y_global % heightmapSize;
	float texels_x = float(patches_x) * 64.0f;
	float texels_y = float(patches_y) * 64.0f;

	uint32_t tx = uv_global.x * texels_x;
	uint32_t ty = uv_global.y * texels_y;
	uint32_t c_00 = sampleColor_nearest_sparse(sparseHeightmapPointers, tx + 0, ty + 0, patches_x, patches_y, heightmapSize, patches);
	uint32_t c_01 = sampleColor_nearest_sparse(sparseHeightmapPointers, tx + 0, ty + 1, patches_x, patches_y, heightmapSize, patches);
	uint32_t c_10 = sampleColor_nearest_sparse(sparseHeightmapPointers, tx + 1, ty + 0, patches_x, patches_y, heightmapSize, patches);
	uint32_t c_11 = sampleColor_nearest_sparse(sparseHeightmapPointers, tx + 1, ty + 1, patches_x, patches_y, heightmapSize, patches);

	uint8_t* rgba_00 = (uint8_t*)&c_00;
	uint8_t* rgba_01 = (uint8_t*)&c_01;
	uint8_t* rgba_10 = (uint8_t*)&c_10;
	uint8_t* rgba_11 = (uint8_t*)&c_11;

	float wx = fmodf(uv_global.x * texels_x, 1.0f);
	float wy = fmodf(uv_global.y * texels_y, 1.0f);

	float w00 = (1.0 - wx) * (1.0 - wy);
	float w10 = (      wx) * (1.0 - wy);
	float w01 = (1.0 - wx) * (      wy);
	float w11 = (      wx) * (      wy);


	uint32_t color = 0;
	uint8_t* rgba = (uint8_t*)&color;

	rgba[0] = rgba_00[0] * w00 + rgba_10[0] * w10 + rgba_01[0] * w01 + rgba_11[0] * w11;
	rgba[1] = rgba_00[1] * w00 + rgba_10[1] * w10 + rgba_01[1] * w01 + rgba_11[1] * w11;
	rgba[2] = rgba_00[2] * w00 + rgba_10[2] * w10 + rgba_01[2] * w01 + rgba_11[2] * w11;
	rgba[3] = rgba_00[3] * w00 + rgba_10[3] * w10 + rgba_01[3] * w01 + rgba_11[3] * w11;

	return color;
}

float sampleHeight(
	SparseHeightmapPointer* sparseHeightmapPointers,
	uint32_t texel_x_global, uint32_t texel_y_global,
	uint32_t patches_x, uint32_t patches_y, uint32_t heightmapSize
){
	int numPatches = patches_x * patches_y;

	float texel_x = fmod(texel_x_global, float(heightmapSize));
	float texel_y = fmod(texel_y_global, float(heightmapSize));

	int texelID = int(texel_x) * heightmapSize + int(texel_y);

	int patch_x = texel_x_global / 64.0;
	int patch_y = texel_y_global / 64.0;
	int patchID = patch_x + patches_x * patch_y;

	if(patchID >= 0 && patchID < numPatches){
		SparseHeightmapPointer ptr = sparseHeightmapPointers[patchID];
		float* heightmap = (float*)ptr.heightmap;

		int texelID = int(texel_x) + heightmapSize * int(texel_y);

		if(heightmap != nullptr & texelID < 64 * 64){
			return heightmap[texelID];
		}

	}

	return 0.0f;
}

extern "C" __global__ 
void kernel_render_patches_triangles(
		const Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		LasTiles lasTiles,
		uint32_t* patchesTriangles,
		TriangleData trianglesData,
		SparseHeightmapPointer* sparseHeightmapPointers
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();


	if (state->patchAsTrianglesQueue > 0) {
	if (uniforms.renderHeightmapsAsPoints) {
		__shared__ int blockPatchIndex;

		state->patchCounter = 0;

		grid.sync();

		while (true) {
			block.sync();

			if (block.thread_rank() == 0) {
				blockPatchIndex = atomicAdd(&state->patchCounter, 1);
			}

			block.sync();

			if (blockPatchIndex >= state->patchAsTrianglesQueue) {
				break;
			}

			Patch& patch = patches.patches[patchesTriangles[blockPatchIndex]];

			processRangeBlock(64 * 64, [&](const size_t pointIndex) {

				Point p;
				p.x = static_cast<float>(pointIndex % 64) * 10.0f + patch.min.x + 5.0f;
				p.y = static_cast<float>(pointIndex / 64) * 10.0f + patch.min.y + 5.0f;
				p.z = patch.heightmap[pointIndex]; // * uniforms.heightmapPatchRadius / uniforms.heightmapNumericalStabilityFactor;
				
				p.color = uniforms.colorHeightmapsByPatch ? colorFromIndex(patch.patchIndex) : 0xffccccccull;

				float u = float(pointIndex % 64) / 64.0f;
				float v = float(pointIndex / 64) / 64.0f;
				uint8_t* rgba = (uint8_t*)&p.color;
				rgba[0] = 255.0f * u;
				rgba[1] = 255.0f * v;
				rgba[2] = 0;
				rgba[3] = 255;

				int texelX = clamp(pointIndex % 64, 0, 63);
				int texelY = clamp(pointIndex / 64, 0, 63);

				uint32_t texelIndex = texelY * uniforms.textureSize + texelX;
				uint32_t texColor = patch.texture[texelIndex];
				p.color = texColor;

				rasterizePoint(p, renderTarget.framebuffer, uniforms.width, uniforms.height, uniforms.transform, uniforms.pointSize);
			});
		}
	} else {
		// add vertex attributes as needed

		struct Vertex {
			// position is the only required attribute and needs to be the position after projection
			vec4 position;
			vec3 worldPos;
			vec3 normal;
			uint32_t patchIndex;
			vec2 uv;
			bool needsFragmentOverlapCheck;
			uint32_t color;
			vec4 fcol;
		};
		rasterizeTrianglesInstanced<Vertex>(
			trianglesData.count,
			state->patchAsTrianglesQueue,
			renderTarget,
			[&](int vertexIndex, int instanceIndex) {
				Vertex vertex;

				auto patchIndex = patchesTriangles[instanceIndex];
				Patch patch = patches.patches[patchIndex];

				vec2 uv = trianglesData.uv[vertexIndex];

				// todo: we could also just compute positions & uvs from vertex indices (we have 64*64*3*2 vertices)
				vec3 position = trianglesData.position[vertexIndex];
				vec3 worldPos = position + vec3{patch.min.x, patch.min.y, 0.0};

				float* heightmap = patch.heightmap;
				int heightmapSize = uniforms.heightmapSize;

				SampleInfo sample = worldPosToSampleInfo(worldPos, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);

				worldPos.z = sampleHeight(sparseHeightmapPointers,
					sample.texel_x_global, sample.texel_y_global,
					uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);

				vertex.worldPos = worldPos;
				
				{ // compute normal
					auto ptr = sparseHeightmapPointers;
					float x_n = sampleHeight(ptr, sample.texel_x_global - 1, sample.texel_y_global - 0, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);
					float x_p = sampleHeight(ptr, sample.texel_x_global + 1, sample.texel_y_global - 0, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);
					float y_n = sampleHeight(ptr, sample.texel_x_global + 0, sample.texel_y_global - 1, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);
					float y_p = sampleHeight(ptr, sample.texel_x_global + 0, sample.texel_y_global + 1, uniforms.patches_x, uniforms.patches_y, uniforms.heightmapSize);

					vec3 dx = {20.0f, 0.0f, x_p - x_n};
					vec3 dy = {0.0f, 20.0f, y_p - y_n};
					vec3 N = normalize(cross(dx, dy));
					vertex.normal = N;
				}

				// uv.x = (vertex.worldPos.x / 10.0f - 1.0f) / float(uniforms.heightmapSize);
				// uv.y = (vertex.worldPos.y / 10.0f - 1.0f) / float(uniforms.heightmapSize);
				// uv.x = (vertex.worldPos.x / 10.0f - 1.0f) / float(uniforms.heightmapSize);
				// uv.y = (texel_y) / float(uniforms.heightmapSize);

				vertex.position = uniforms.transform * vec4{worldPos.x, worldPos.y, worldPos.z, 1.0f};
				vertex.patchIndex = patch.patchIndex;
				vertex.uv = sample.uv_global;
				vertex.color = sampleColor_nearest_sparse(
					sparseHeightmapPointers, 
					sample.texel_x_global, sample.texel_y_global,
					uniforms.patches_x, uniforms.patches_y, 
					uniforms.heightmapSize, patches
				);
				vertex.fcol = vec4{
					(vertex.color >>  0) & 0xff,
					(vertex.color >>  8) & 0xff,
					(vertex.color >> 16) & 0xff,
					(vertex.color >> 24) & 0xff,
				};
				// vertex.color = sampleColor_linear_sparse(
				// 	sparseHeightmapPointers, 
				// 	sample,
				// 	uniforms.patches_x, uniforms.patches_y, 
				// 	uniforms.heightmapSize, patches
				// );

				// vertex.needsFragmentOverlapCheck = !patch.noOverlaps;

				return vertex;
			},
			[&](const Fragment& fragment, const Vertex& v0, const Vertex& v1, const Vertex& v2) {
				


				// vec3 normal = normalize(cross(v1.worldPos - v0.worldPos, v2.worldPos - v1.worldPos));
				vec3 normal = fragment.interpolateAttribute(v0.normal, v1.normal, v2.normal);
				vec3 light = normalize(vec3{1.0f, 0.0f, 1.0f});
				float diffuse = max(dot(normal, light), 0.0f);
				diffuse = clamp(diffuse, 0.1f, 1.0f);
				diffuse = 1.0f;

				uint32_t albedo = uniforms.colorHeightmapsByPatch ? colorFromIndex(v0.patchIndex) : 0xffccccccull;

				vec2 uv = fragment.interpolateAttribute(v0.uv, v1.uv, v2.uv);

				if (v0.color != 0) {
					albedo = v0.color;
				}

				if(uniforms.disableTextures){
					albedo = 0xffffffff;
				}

				uint32_t color = albedo;
				auto rgba = (uint8_t*)&color;
				rgba[0] = static_cast<uint8_t>(static_cast<float>(rgba[0]) * diffuse);
				rgba[1] = static_cast<uint8_t>(static_cast<float>(rgba[1]) * diffuse);
				rgba[2] = static_cast<uint8_t>(static_cast<float>(rgba[2]) * diffuse);

				uint32_t texel_x_global = uv.x * uniforms.patches_x * 64;
				uint32_t texel_y_global = uv.y * uniforms.patches_y * 64;

				color = sampleColor_linear_sparse(
					sparseHeightmapPointers, 
					uv,
					uniforms.patches_x, uniforms.patches_y, 
					uniforms.heightmapSize, patches
				);

				
				

				if(uniforms.disableTextures){
					vec2 uv = fragment.interpolateAttribute(v0.uv, v1.uv, v2.uv);
					rgba[0] = 200.0f * diffuse;
					rgba[1] = 200.0f * diffuse;
					rgba[2] = 200.0f * diffuse;
				}else{
					rgba[0] = rgba[0] * clamp(1.5f * diffuse, 0.7f, 1.0f);
					rgba[1] = rgba[1] * clamp(1.5f * diffuse, 0.7f, 1.0f);
					rgba[2] = rgba[2] * clamp(1.5f * diffuse, 0.7f, 1.0f);
				}

				if((color >> 24) < 150){
					color = 0;
				}

				// discard fragments overlapping with high res chunks
				// if (v0.needsFragmentOverlapCheck) 
				// {
				// 	const vec3 fragWorldPos = fragment.interpolateAttribute(v0.worldPos, v1.worldPos, v2.worldPos);
				// 	for (size_t i = 0; i < state->numHighlyVisibleLasTiles; ++i) {
				// 		const Tile& tile = lasTiles.tiles[state->highlyVisibleLasTileIds[i]];

				// 		if (
				// 			tile.min.x <= fragWorldPos.x - 5.0f 
				// 			&& tile.max.x >= fragWorldPos.x  + 10.0f
				// 			&& tile.min.y <= fragWorldPos.y - 5.0f
				// 			&& tile.max.y >= fragWorldPos.y + 10.0f
				// 		) {
				// 			// return DISCARD_FRAGMENT;
				// 			color = 0;
				// 		}
				// 	}
				// }

				// if(patches.patches[sparseHeightmapPointers->patchIndex].isVisible){
				// 	color = 0;
				// }
				

				// vec4 fcol = fragment.interpolateAttribute(v0.fcol, v1.fcol, v2.fcol);

				// rgba[0] = fcol[0];
				// rgba[1] = fcol[1];
				// rgba[2] = fcol[2];
				// rgba[3] = fcol[3];

				// rgba[0] = 255.0f * normal.x;
				// rgba[1] = 255.0f * normal.y;
				// rgba[2] = 255.0f * normal.z;

				// { // DRAW UV
				// 	vec2 uv = fragment.interpolateAttribute(v0.uv, v1.uv, v2.uv);
				// 	rgba[0] = 255.0f * uv.x;
				// 	rgba[1] = 255.0f * uv.y;
				// 	rgba[2] = 0.0f;
				// 	rgba[3] = 255.0f;
				// }

				// {
				// 	vec3 N = fragment.interpolateAttribute(v0.normal, v1.normal, v2.normal);

				// 	// vec3 N = v0.normal * fragment.s + v1.normal * fragment.t + v2.normal * fragment.v;
				// 	// N = v0.normal * fragment.s + v1.normal * fragment.v + v2.normal * fragment.t;
				// 	// N = v0.normal * fragment.t + v1.normal * fragment.v + v2.normal * fragment.s;
				// 	// N = v0.normal * fragment.v + v1.normal * fragment.s + v2.normal * fragment.t;

				// 	rgba[0] = 255.0f * N.x;
				// 	rgba[1] = 255.0f * N.y;
				// 	rgba[2] = 255.0f * N.z;
				// }


				return color;
			});
		}
	}

}


extern "C" __global__ 
void kernel_render_patches_points(
		const Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		const Patches patches,
		uint32_t* patchesPoints
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	if (state->patchAsPointsQueue > 0) {
		__shared__ int blockPatchIndex;

		state->patchCounter = 0;

		grid.sync();

		while (true) {
			block.sync();

			if (block.thread_rank() == 0) {
				blockPatchIndex = atomicAdd(&state->patchCounter, 1);
			}

			block.sync();

			if (blockPatchIndex >= state->patchAsPointsQueue) {
				break;
			}

			Patch& patch = patches.patches[patchesPoints[blockPatchIndex]];
			processRangeBlock(patch.numPoints, [&](const size_t pointIndex) {
				Point p = patch.points[pointIndex];
				if (uniforms.colorChunkPointsByPatch) {
					p.color = colorFromIndex(patch.patchIndex);
				}
				
				// if(pointIndex % 5 == 0)
				rasterizePoint(p, renderTarget.framebuffer, uniforms.width, uniforms.height, uniforms.transform, uniforms.pointSize);
			});
		}
	}

}

__device__ Lines g_lines;
__device__ Vertex g_vertices[20'000'000];

extern "C" __global__ 
void kernel_render_boundingboxes(
		const Uniforms uniforms,
		RenderTarget renderTarget,
		Stats* stats,
		DeviceState* state,
		Box3* boxes,
		uint32_t* colors,
		uint32_t numBoxes
){
	auto grid = cg::this_grid();
	auto block = cg::this_thread_block();

	if(grid.thread_rank() == 0){
		// printf("numBoxes: %d \n", numBoxes);

		// Box3 box = boxes[0];
		// printf("%.1f, %.1f, %.1f \n", 
		// 	box.min.x, 
		// 	box.min.y, 
		// 	box.min.z
		// );
		// printf("%.1f, %.1f, %.1f \n", 
		// 	uniforms.boxMin.x, 
		// 	uniforms.boxMin.y, 
		// 	uniforms.boxMin.z
		// );

	}

	grid.sync();

	Lines* lines = &g_lines;
	lines->count = 0;
	lines->vertices = &g_vertices[0];

	grid.sync();

	processRange(numBoxes, [&](int index){
		Box3 box = boxes[index];

		vec3 pos = {
			(box.min.x + box.max.x) / 2.0f,
			(box.min.y + box.max.y) / 2.0f,
			(box.min.z + box.max.z) / 2.0f,
		};
		vec3 size = {
			(box.max.x - box.min.x),
			(box.max.y - box.min.y),
			(box.max.z - box.min.z),
		};

		uint32_t color = colors[index];

		drawBoundingBox(lines, pos, size, color);
	});

	grid.sync();

	// if(grid.thread_rank() == 0) printf("count: %d \n", numBoxes);

	mat4 worldViewProj = renderTarget.proj * renderTarget.view;
	rasterizeLines(lines, renderTarget.framebuffer, renderTarget.width, renderTarget.height, worldViewProj);

}



extern "C" __global__ 
void kernel_toOpenGL(
	const Uniforms uniforms,
	RenderTarget renderTarget,
	hipSurfaceObject_t gl_colorbuffer
){
	processRange(0, uniforms.width * uniforms.height, [&](int pixelIndex) {
		int x = pixelIndex % int(uniforms.width);
		int y = pixelIndex / int(uniforms.width);

		uint64_t encoded = renderTarget.framebuffer[pixelIndex];
		uint32_t color = encoded & 0xffffffffull;

		surf2Dwrite(color, gl_colorbuffer, x * 4, y);
	});
}